#include "hip/hip_runtime.h"
#include "nms_cuda.h"

__device__ int get_index(){
  int blockId=blockIdx.y*gridDim.x+blockIdx.x;
  int threadId=blockId*blockDim.x+threadIdx.x;
  return threadId;
}

__device__ int get_block_prefix(){
    int blockId=blockIdx.y*gridDim.x+blockIdx.x;
    int block_prefix=blockId*blockDim.x;
    return block_prefix;
}

__device__ int get_thread(){
  return threadIdx.x;
}

__global__ void nms_cuda_imp(long* bbox, int64_t* bbox_size,
   float* mask,int64_t* mask_size,float thresh){
     int index=get_index();
     int block_prefix=get_block_prefix();
     for(int i=0;i<bbox_size[1];++i){
       __syncthreads();
       if(index>=bbox_size[0]*bbox_size[1]){
         continue;
       }
       if(i>=get_thread()){
         continue;
       }
       if(mask[block_prefix+i]==0){
         continue;
       }
       if(mask[index]==0){
         continue;
       }
       long x11=bbox[4*(block_prefix)+0];
       long y11=bbox[4*(block_prefix)+1];
       long x12=bbox[4*(block_prefix)+2];
       long y12=bbox[4*(block_prefix)+3];
       long x21=bbox[4*(index)+0];
       long y21=bbox[4*(index)+1];
       long x22=bbox[4*(index)+2];
       long y22=bbox[4*(index)+3];
       int areas_u=(x12-x11)*(y12-y11)+(x22-x21)*(y22-y21);
       int max_x1=((x11>=x21)?x11:x21);
       int max_y1=((y11>=y21)?y11:y21);
       int min_x2=((x12>=x22)?x12:x22);
       int min_y2=((y12>=y22)?y12:y22);
       int w=min_x2-max_x1;
       w=(w>=0?w:0);
       int h=min_y2-max_y1;
       h=(h>=0?h:0);
       int areas_n=w*h;
       if(areas_u-areas_n==0){
         continue;
       }
       if(float(areas_n)/float(areas_u-areas_n)>thresh){
         mask[index]=0;
       }
     }

   }

void cuda_cpy(int64_t* from,int64_t** to,int size){
  hipMalloc((void**)to,size*sizeof(int64_t));
  hipMemcpy(*to,from,size*sizeof(int64_t),hipMemcpyHostToDevice);
}

void nms_cuda(long* bbox, int64_t* bbox_size,
   float* mask,int64_t* mask_size,float thresh,hipStream_t stream){
  int d1=1;
  int d2=1;
  if(bbox_size[0]>512){
    d2=(bbox_size[0]+511)/512;
    d1=512;
  }else{
    d1=bbox_size[0];
    d2=1;
  }
  dim3 batch(d1,d2,1);
  dim3 thread(bbox_size[1],1,1);
  int64_t* bbox_size_cuda;
  int64_t* mask_size_cuda;
  cuda_cpy(bbox_size,&bbox_size_cuda,3);
  cuda_cpy(mask_size,&mask_size_cuda,2);
  nms_cuda_imp<<<batch,thread,0,stream>>>(bbox,bbox_size_cuda,mask,mask_size_cuda,thresh);

  hipFree(bbox_size_cuda);
  hipFree(mask_size_cuda);
}
